#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <random>

hipError_t grayScaleWithCuda(uint8_t* inputImage, uint8_t* outputImage, int imageWidth, int imageHeight, int imageChannels);
void grayScaleSerial(uint8_t* inputImage, uint8_t* outputImage, int imageWidth, int imageHeight, int imageChannels);

__global__ void RGBtoGrayScale(uint8_t* ucharImg, uint8_t* grayImg, int width, int height) {
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	int row = blockDim.y * blockIdx.y + threadIdx.y;

	if (col < width && row < height) {
		int idx = width * row + col;
		uint8_t r = ucharImg[3 * idx];
		uint8_t g = ucharImg[3 * idx + 1];
		uint8_t b = ucharImg[3 * idx + 2];
		grayImg[idx] = (uint8_t)(0.21 * r + 0.71 * g + 0.07 * b);
	}
}

int main() {
	int imageWidth = 1920;
	int imageHeight = 1080;
	int imageChannels = 3;

	uint8_t* inputImage = new uint8_t[imageWidth * imageHeight * imageChannels];
	uint8_t* outputImage = new uint8_t[imageWidth * imageHeight];
	uint8_t* outputImageSerial = new uint8_t[imageWidth * imageHeight];

	// Randomly generate input image
	srand(time(NULL));
	for (int i = 0; i < imageWidth * imageHeight * imageChannels; i++) {
		inputImage[i] = (uint8_t)(rand() / RAND_MAX);
	}
	printf("Finished Generating Random Input Image.\n");

	// Serial
	grayScaleSerial(inputImage, outputImageSerial, imageWidth, imageHeight, imageChannels);
	printf("Finished Serial Image Grayscale.\n");

	// Parallel transfer image to grayscale
	hipError_t cudaStatus = grayScaleWithCuda(inputImage, outputImage, imageWidth, imageHeight, imageChannels);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "grayScaleWithCuda failed!");
		return 1;
	}
	printf("Finished Parallel Image Grayscale.\n");

	// check if correct
	bool correct = true;
	for (int i = 0; i < imageWidth * imageHeight; i++) {
		if (abs(outputImageSerial[i] - outputImage[i]) > 0.001) {
			correct = false;
		}
	}
	if (correct) printf("Correct Result!\n");
	else printf("Wrong Result!\n");

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	delete(inputImage);
	delete(outputImage);
	delete(outputImageSerial);

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t grayScaleWithCuda(uint8_t* inputImage, uint8_t* outputImage, int imageWidth, int imageHeight, int imageChannels) {
	uint8_t* dev_inImg = 0;
	uint8_t* dev_outImg = 0;
	hipError_t cudaStatus;

	// Allocate GPU memory for input image
	cudaStatus = hipMalloc((void**)&dev_inImg, imageWidth * imageHeight * imageChannels * sizeof(uint8_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Allocate GPU memory for output image
	cudaStatus = hipMalloc((void**)&dev_outImg, imageWidth * imageHeight * sizeof(uint8_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input image from host to device.
	cudaStatus = hipMemcpy(dev_inImg, inputImage, imageWidth * imageHeight * imageChannels * sizeof(uint8_t), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel
	dim3 dimBlock(32, 32, 1);
	dim3 dimGrid(ceil(1.0 * imageWidth / 32.0), ceil(1.0 * imageHeight / 32.0), 1);
	RGBtoGrayScale<<<dimGrid, dimBlock>>>(dev_inImg, dev_outImg, imageWidth, imageHeight);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "RGBtoGrayScale launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching RGBtoGrayScale!\n", cudaStatus);
		goto Error;
	}

	// Copy output image from device to host.
	cudaStatus = hipMemcpy(outputImage, dev_outImg, imageWidth * imageHeight * sizeof(uint8_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_inImg);
	hipFree(dev_outImg);

	return cudaStatus;
}

void grayScaleSerial(uint8_t* inputImage, uint8_t* outputImage, int imageWidth, int imageHeight, int imageChannels) {
	for (int i = 0; i < imageHeight; i++) {
		for (int j = 0; j < imageWidth; j++) {
			int idx = i * imageWidth + j;
			uint8_t r = inputImage[3 * idx];
			uint8_t	g = inputImage[3 * idx + 1];
			uint8_t b = inputImage[3 * idx + 2];
			outputImage[idx] = (uint8_t)(0.21 * r + 0.71 * g + 0.07 * b);
		}
	}
}