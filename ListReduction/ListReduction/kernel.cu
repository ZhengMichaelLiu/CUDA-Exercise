#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <time.h>
#include <random>

#define BLOCK_SIZE 512

hipError_t listReductionWithCuda(int* inputArray, int* outputArray, int inputSize, int outputSize);

__global__ void listReductionKernel(int* inputArray, int* outputArray, int inputSize) {

	// use one block to process 2 * BLOCK_SIZE elements
	__shared__ int partialSum[2 * BLOCK_SIZE];

	int i = threadIdx.x;
	int start = 2 * blockDim.x * blockIdx.x;

	// load 2 * BLOCK_SIZE elements in
	if (start + i < inputSize) {
		partialSum[i] = inputArray[start + i];
	} else {
		partialSum[i] = 0;
	}

	if (start + blockDim.x + i < inputSize) {
		partialSum[blockDim.x + i] = inputArray[start + blockDim.x + i];
	} else {
		partialSum[blockDim.x + i] = 0;
	}

	// parallel reduction
	for (int stride = blockDim.x; stride >= 1; stride /= 2) {
		__syncthreads();
		if (i < stride) {
			partialSum[i] += partialSum[i + stride];
		}
	}
	// write result to output
	if (i == 0) {
		outputArray[blockIdx.x] = partialSum[0];
	}
}

int main() {

	int inputArraySize = 45635231;
	int* inputArray = new int[inputArraySize];

	int outputArraySize = inputArraySize / (BLOCK_SIZE * 2);
	if (inputArraySize % (BLOCK_SIZE * 2)) {
		outputArraySize++;
	}
	int* outputArray = new int[outputArraySize];

	// Randomly Generate input array
	srand(time(NULL));
	for (int i = 0; i < inputArraySize; i++) {
		inputArray[i] = rand() % 3;
	}
	printf("Finished Generating Input Array.\n");

	// List Reduction in serial
	clock_t start = clock();
	int serialResult = 0;
	for (int i = 0; i < inputArraySize; i++) {
		serialResult += inputArray[i];
	}
	clock_t end = clock();
	double elapsed = double(end - start) / CLOCKS_PER_SEC;
	printf("Finished Calculating List Reduction in Serial: %d.\nTime Elapsed: %f seconds.\n", serialResult, elapsed);

	// List Reduction in parallel.
	hipError_t cudaStatus = listReductionWithCuda(inputArray, outputArray, inputArraySize, outputArraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "listReductionWithCuda failed!");
		return 1;
	}

	int parallelResult = 0;
	for (int i = 0; i < outputArraySize; i++) {
		parallelResult += outputArray[i];
	}
	printf("Finished Calculating List Reduction in Parallel. %d\n", parallelResult);

	if (abs(parallelResult - serialResult) > 0.001) {
		printf("Wrong Result!\n");
	} else {
		printf("Correct Result!\n");
	}
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	delete(inputArray);
	delete(outputArray);

	return 0;
}

// List Reduction in parallel.
hipError_t listReductionWithCuda(int* inputArray, int* outputArray, int inputSize, int outputSize) {
	int* dev_in = 0;
	int* dev_out = 0;
	hipError_t cudaStatus;

	// Allocate device memory for input list
	cudaStatus = hipMalloc((void**)&dev_in, inputSize * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Allocate device memory for output list
	cudaStatus = hipMalloc((void**)&dev_out, outputSize * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input list from host to device.
	cudaStatus = hipMemcpy(dev_in, inputArray, inputSize * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel
	dim3 dimGrid(outputSize, 1, 1);
	dim3 dimBlock(BLOCK_SIZE, 1, 1);
	listReductionKernel<<<dimGrid, dimBlock>>>(dev_in, dev_out, inputSize);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "listReductionKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching listReductionKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from device to host.
	cudaStatus = hipMemcpy(outputArray, dev_out, outputSize * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_in);
	hipFree(dev_out);

	return cudaStatus;
}